#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2020-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 * Copyright (C) 2019-2019  Mael Keryell
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <stdio.h>
#include <stdint.h>
#include <starpu.h>

#define THREADS_PER_BLOCK 64

static inline long long jlstarpu_max(long long a, long long b)
{
	return (a > b) ? a : b;
}

static inline long long jlstarpu_interval_size(long long start, long long step, long long stop)
{
    if (stop >= start){
            return jlstarpu_max(0, (stop - start + 1) / step);
    } else {
            return jlstarpu_max(0, (stop - start - 1) / step);
    }
}


__device__ static inline long long jlstarpu_max__device(long long a, long long b)
{
	return (a > b) ? a : b;
}

__device__ static inline long long jlstarpu_interval_size__device(long long start, long long step, long long stop)
{
	if (stop >= start){
		return jlstarpu_max__device(0, (stop - start + 1) / step);
	} else {
		return jlstarpu_max__device(0, (stop - start - 1) / step);
	}
}


__global__ void mandelbrot(int64_t kernel_ids__start_1, int64_t kernel_ids__step_1, int64_t kernel_ids__dim_1, int64_t kernel_ids__start_2, 
                           int64_t kernel_ids__step_2, int64_t kernel_ids__dim_2, double* ptr_hF6lCYyJ, int64_t local_width, 
                           int64_t* ptr_qoUGBRtY, int64_t local_height, double conv_limit, int64_t* ptr_A5zD9sJZ, 
                           uint32_t ld_A5zD9sJZ)
{
    int64_t THREAD_ID = (int64_t) ((((blockIdx).x) * ((blockDim).x)) + ((threadIdx).x));
    
    if ((THREAD_ID) >= (((1) * (kernel_ids__dim_2)) * (kernel_ids__dim_1)))
    {
        return ;
    };
    int64_t kernel_ids__index_1 = (int64_t) (((THREAD_ID) / ((1) * (kernel_ids__dim_2))) % (kernel_ids__dim_1));
    int64_t kernel_ids__index_2 = (int64_t) (((THREAD_ID) / (1)) % (kernel_ids__dim_2));
    int64_t x = (int64_t) ((kernel_ids__start_1) + ((kernel_ids__index_1) * (kernel_ids__step_1)));
    int64_t y = (int64_t) ((kernel_ids__start_2) + ((kernel_ids__index_2) * (kernel_ids__step_2)));
    double max_iterations = (double) (ptr_hF6lCYyJ[(5) - (1)]);
    double zoom = (double) ((ptr_hF6lCYyJ[(3) - (1)]) * (0.25296875));
    int64_t X = (int64_t) ((x) + ((local_width) * ((ptr_qoUGBRtY[(2) - (1)]) - (1))));
    int64_t Y = (int64_t) ((y) + ((local_height) * ((ptr_qoUGBRtY[(1) - (1)]) - (1))));
    double cr = (double) ((ptr_hF6lCYyJ[(1) - (1)]) + (((X) - ((ptr_hF6lCYyJ[(3) - (1)]) / (2))) / (zoom)));
    double zr = (double) (cr);
    double ci = (double) ((ptr_hF6lCYyJ[(2) - (1)]) + (((Y) - ((ptr_hF6lCYyJ[(4) - (1)]) / (2))) / (zoom)));
    double zi = (double) (ci);
    int64_t n = (int64_t) (0);
    int64_t b1 = (int64_t) (((n) < (max_iterations)) + ((((zr) * (zr)) + ((zi) * (zi))) < ((conv_limit) * (conv_limit))));
    
    while ((b1) >= (2))
    {
        double tmp = (double) ((((zr) * (zr)) - ((zi) * (zi))) + (cr));
        zi = ((2) * (zr) * (zi)) + (ci);
        zr = tmp;
        n = (n) + (1);
        b1 = ((n) <= (max_iterations)) + ((((zr) * (zr)) + ((zi) * (zi))) <= ((conv_limit) * (conv_limit)));
    }
    ;
    
    if ((n) < (max_iterations))
    {
        ptr_A5zD9sJZ[((y) + (((x) - (1)) * (ld_A5zD9sJZ))) - (1)] = ((255) * (n)) / (max_iterations);
    } else
    {
        ptr_A5zD9sJZ[((y) + (((x) - (1)) * (ld_A5zD9sJZ))) - (1)] = 0;
    }
    ;
}



extern "C" void CUDA_mandelbrot(void** buffers_uwrYFDVe, void* cl_arg_uwrYFDVe)
{
    uint32_t ld_A5zD9sJZ = (uint32_t) (STARPU_MATRIX_GET_LD(buffers_uwrYFDVe[(1) - (1)]));
    int64_t* ptr_A5zD9sJZ = (int64_t*) (STARPU_MATRIX_GET_PTR(buffers_uwrYFDVe[(1) - (1)]));
    double* ptr_hF6lCYyJ = (double*) (STARPU_VECTOR_GET_PTR(buffers_uwrYFDVe[(2) - (1)]));
    int64_t* ptr_qoUGBRtY = (int64_t*) (STARPU_VECTOR_GET_PTR(buffers_uwrYFDVe[(3) - (1)]));
    int64_t local_width = (int64_t) (STARPU_MATRIX_GET_NY(buffers_uwrYFDVe[(1) - (1)]));
    int64_t local_height = (int64_t) (STARPU_MATRIX_GET_NX(buffers_uwrYFDVe[(1) - (1)]));
    double conv_limit = (double) (2.0);
    int64_t kernel_ids__start_1 = (int64_t) (1);
    int64_t kernel_ids__step_1 = (int64_t) (1);
    int64_t kernel_ids__dim_1 = (int64_t) (jlstarpu_interval_size(kernel_ids__start_1, kernel_ids__step_1, local_width));
    int64_t kernel_ids__start_2 = (int64_t) (1);
    int64_t kernel_ids__step_2 = (int64_t) (1);
    int64_t kernel_ids__dim_2 = (int64_t) (jlstarpu_interval_size(kernel_ids__start_2, kernel_ids__step_2, local_height));
    int64_t nthreads = (int64_t) (((1) * (kernel_ids__dim_1)) * (kernel_ids__dim_2));
    int64_t nblocks = (int64_t) ((((nthreads) + (THREADS_PER_BLOCK)) - (1)) / (THREADS_PER_BLOCK));
    
    mandelbrot
        <<< nblocks, THREADS_PER_BLOCK, 0, starpu_cuda_get_local_stream()
        >>> (kernel_ids__start_1, kernel_ids__step_1, kernel_ids__dim_1, kernel_ids__start_2, 
             kernel_ids__step_2, kernel_ids__dim_2, ptr_hF6lCYyJ, local_width, 
             ptr_qoUGBRtY, local_height, conv_limit, ptr_A5zD9sJZ, 
             ld_A5zD9sJZ);
    ;
    hipError_t status = hipGetLastError();
    if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
    hipStreamSynchronize(starpu_cuda_get_local_stream());
}


