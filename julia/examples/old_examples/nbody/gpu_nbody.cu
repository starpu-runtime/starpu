#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2019-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 * Copyright (C) 2019-2019  Mael Keryell
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <starpu.h>

struct Params
{
  unsigned taskx;
  unsigned epsilon;
};

__global__ void gpuNbodyKernel(double *P, double *subA, double *M,
			     size_t nxP, size_t nxA, size_t nxM,
			     size_t ldP, size_t ldA,
			     struct Params params)
{
  size_t id, i, j, k;
  double dx, dy, modul;

  id = blockIdx.x * blockDim.x + threadIdx.x;
  i = id % nxA;
  j = id / nxA;

  if (j >= 1){
    return;
  }

  double sumaccx;
  double sumaccy;
  
  for (k = 0; k < nxP; k++){
    if (k != id + nxA*params.taskx){
      dx = P[k] - P[id + nxA*params.taskx];
      dy = P[k + ldP] - P[id + nxA*params.taskx + ldP];
      
      modul = dx * dx + dy * dy;

      sumaccx = 6.674e-11 * M[k] * dx / pow(modul + params.epsilon, 3);
      sumaccy = 6.674e-11 * M[k] * dy / pow(modul + params.epsilon, 3);
    }
  }
 
  subA[i] = sumaccx;
  subA[i + ldA] = sumaccy;

  // P[id + nxA * params.taskx] = subA[i];

  // subA[i] = 0;
  // subA[i + ldA] = 1;
  
}

#define THREADS_PER_BLOCK 64

extern "C" void gpu_nbody(void * descr[], void * args)
{

  double *d_P, *d_subA, *d_M;
  size_t nxP, nxA, nxM;
  size_t ldA, ldP;
  size_t nblocks;

  struct Params *params = (struct Params *) args;

  d_P = (double *) STARPU_MATRIX_GET_PTR(descr[0]);
  d_subA = (double *) STARPU_MATRIX_GET_PTR(descr[1]);
  d_M = (double *) STARPU_MATRIX_GET_PTR(descr[2]);

  nxP = STARPU_MATRIX_GET_NX(descr[0]);
  nxA = STARPU_MATRIX_GET_NX(descr[1]);
  nxM = STARPU_MATRIX_GET_NX(descr[2]);

  ldP = STARPU_MATRIX_GET_LD(descr[0]);
  ldA = STARPU_MATRIX_GET_LD(descr[1]);

  nblocks = (nxA + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  gpuNbodyKernel
    <<< nblocks, THREADS_PER_BLOCK, 0, starpu_cuda_get_local_stream()
    >>> (d_P,  d_subA, d_M, nxP, nxA, nxM, ldP, ldA, *params);
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);

  hipStreamSynchronize(starpu_cuda_get_local_stream());

}







__global__ void gpuNbody2Kernel(double *d_subP, double *d_subV, double *d_subA,
			      size_t nxP, size_t nxV, size_t nxA,
			      size_t ldP, size_t ldV, size_t ldA,
			      struct Params params)
{

  size_t id, i, j;

  id = blockIdx.x * blockDim.x + threadIdx.x;

  i = id % nxP;
  j = id / nxP;

  if (j >= 1){
    return;
  }

  d_subV[i] = d_subV[i] + 3600*d_subA[i];
  d_subV[i + ldV] = d_subV[i + ldV] + 3600*d_subA[i + ldA];

  d_subP[i] = d_subP[i] + 3600*d_subV[i];
  d_subP[i + ldP] = d_subP[i + ldP] + 3600*d_subV[i + ldV];
}


extern "C" void gpu_nbody2(void * descr[], void *args)
{
  double *d_subP, *d_subV, *d_subA;
  size_t nxP, nxV, nxA;
  size_t ldP, ldV, ldA;
  size_t nblocks;

  struct Params *params = (struct Params *) args;

  d_subP = (double *) STARPU_MATRIX_GET_PTR(descr[0]);
  d_subV = (double *) STARPU_MATRIX_GET_PTR(descr[1]);
  d_subA = (double *) STARPU_MATRIX_GET_PTR(descr[2]);

  nxP = STARPU_MATRIX_GET_NX(descr[0]);
  nxV = STARPU_MATRIX_GET_NX(descr[1]);
  nxA = STARPU_MATRIX_GET_NX(descr[2]);

  ldP = STARPU_MATRIX_GET_LD(descr[0]);
  ldV = STARPU_MATRIX_GET_LD(descr[1]);
  ldA = STARPU_MATRIX_GET_LD(descr[2]);

  nblocks = (nxA + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  gpuNbody2Kernel
    <<< nblocks, THREADS_PER_BLOCK, 0, starpu_cuda_get_local_stream()
    >>> (d_subP, d_subV, d_subA, nxP, nxV, nxA, ldP, ldV, ldA, *params);
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);

  hipStreamSynchronize(starpu_cuda_get_local_stream());
}
