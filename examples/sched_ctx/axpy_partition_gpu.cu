#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2016-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 * Copyright (C) 2016-2016  Uppsala University
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/*
 * This creates two dumb vectors, splits them into chunks, and for each pair of
 * chunk, run axpy on them.
 */

#include <starpu.h>
#include "axpy_partition_gpu.h"
#include <stdio.h>

//This code demonstrates how to transform a kernel to execute on a given set of GPU SMs.


// Original kernel
__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n)  y[i] = a*x[i] + y[i];
}




// Transformed kernel
__global__ void saxpy_partitioned(__P_KARGS, int n, float a, float *x, float *y)
{
  __P_BEGIN;
  __P_LOOPX;
        int i = blockid.x*blockDim.x + threadIdx.x; // note that blockIdx is replaced.
	if (i<n)  y[i] = a*x[i] + y[i];
  __P_LOOPEND;
}
      

extern "C" void cuda_axpy(void *descr[], void *_args)
{
	 float a = *((float *)_args);

        unsigned n = STARPU_VECTOR_GET_NX(descr[0]);

        float *x = (float *)STARPU_VECTOR_GET_PTR(descr[0]);
        float *y = (float *)STARPU_VECTOR_GET_PTR(descr[1]);

	int SM_mapping_start = -1;
	int SM_mapping_end = -1; 
  	int SM_allocation = -1;
  
	hipStream_t stream = starpu_cuda_get_local_stream();
	int workerid = starpu_worker_get_id();
    	starpu_sched_ctx_get_sms_interval(workerid, &SM_mapping_start, &SM_mapping_end);
	SM_allocation = SM_mapping_end - SM_mapping_start;
	int dimensions = 512;	
	//partitioning setup
//	int SM_mapping_start = 0;
//  	int SM_allocation = 13;
  
	__P_HOSTSETUP(saxpy_partitioned,dim3(dimensions,1,1),dimensions,0,SM_mapping_start,SM_allocation,stream);

  	saxpy_partitioned<<<width,dimensions,0,stream>>>(__P_HKARGS,n,a,x,y);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
