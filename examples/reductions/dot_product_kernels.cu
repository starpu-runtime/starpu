#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2009-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/* Trivial dot reduction CUDA kernel */

#include <starpu.h>

#define DOT_TYPE double

static __global__ void cuda_redux(DOT_TYPE *dota, DOT_TYPE *dotb)
{
	*dota = *dota + *dotb;
	return;
}

extern "C" void redux_cuda_func(void *descr[], void *_args)
{
	(void)_args;
	DOT_TYPE *dota = (DOT_TYPE *)STARPU_VARIABLE_GET_PTR(descr[0]);
	DOT_TYPE *dotb = (DOT_TYPE *)STARPU_VARIABLE_GET_PTR(descr[1]);

	cuda_redux<<<1,1, 0, starpu_cuda_get_local_stream()>>>(dota, dotb);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
