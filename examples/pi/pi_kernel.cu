#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/* First draw a series of coordinates, then count how many fall inside the
 * circle quarter */

#include "SobolQRNG/sobol_gpu.h"
#include "pi.h"

#define MAXNBLOCKS	128
#define MAXTHREADSPERBLOCK	256

static __global__ void monte_carlo(TYPE *random_numbers_x, TYPE *random_numbers_y,
						unsigned n, unsigned *output_cnt)
{
	__shared__ unsigned scnt[MAXTHREADSPERBLOCK];

	/* Do we have a successful shot ? */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;

	const int nthreads = gridDim.x * blockDim.x;

	/* Blank the shared mem buffer */
	if (threadIdx.x < MAXTHREADSPERBLOCK)
		scnt[threadIdx.x] = 0;

	__syncthreads();
	int ind;
	for (ind = tid; ind < n; ind += nthreads)
	{
		TYPE x = random_numbers_x[ind];
		TYPE y = random_numbers_y[ind];
		TYPE dist = (x*x + y*y);

		unsigned success = (dist <= 1.0f)?1:0;

		scnt[threadIdx.x] += success;

	}

	__syncthreads();

	/* Perform a reduction to compute the sum on each thread within that block */

	/* NB: We assume that the number of threads per block is a power of 2 ! */
	unsigned s;
	for (s = blockDim.x/2; s!=0; s>>=1)
	{
		if (threadIdx.x < s)
			scnt[threadIdx.x] += scnt[threadIdx.x + s];

		__syncthreads();
	}

	/* report the number of successful shots in the block */
	if (threadIdx.x == 0)
		output_cnt[blockIdx.x] = scnt[0];

	__syncthreads();
}

static __global__ void sum_per_block_cnt(unsigned *output_cnt, unsigned *cnt)
{
	__shared__ unsigned accumulator[MAXNBLOCKS];

	unsigned i;

	/* Load the values from global mem */
	for (i = 0; i < blockDim.x; i++)
		accumulator[i] = output_cnt[i];

	__syncthreads();

	/* Perform a reduction in shared memory */
	unsigned s;
	for (s = blockDim.x/2; s!=0; s>>=1)
	{
		if (threadIdx.x < s)
			accumulator[threadIdx.x] += accumulator[threadIdx.x + s];

		__syncthreads();
	}

	/* Save the result in global memory */
	if (threadIdx.x == 0)
		*cnt = accumulator[0];
}

extern "C" void cuda_kernel(void *descr[], void *cl_arg)
{
	hipError_t cures;

	unsigned *directions = (unsigned *)STARPU_VECTOR_GET_PTR(descr[0]);
	unsigned long long *nshot_per_task = (unsigned long long *) cl_arg;
	unsigned nx = *nshot_per_task;

	/* Generate Random numbers */
	float *random_numbers;
	hipMalloc((void **)&random_numbers, 2*nx*sizeof(float));
	STARPU_ASSERT(random_numbers);

	sobolGPU(2*nx/n_dimensions, n_dimensions, directions, random_numbers);
	hipStreamSynchronize(starpu_cuda_get_local_stream());

	TYPE *random_numbers_x = &random_numbers[0];
	TYPE *random_numbers_y = &random_numbers[nx];

	unsigned *cnt = (unsigned *)STARPU_VECTOR_GET_PTR(descr[1]);

	/* How many blocks do we use ? */
	unsigned nblocks = 128; // TODO

	STARPU_ASSERT(nblocks <= MAXNBLOCKS);

	unsigned *per_block_cnt;
	hipMalloc((void **)&per_block_cnt, nblocks*sizeof(unsigned));

	STARPU_ASSERT((nx % nblocks) == 0);

	/* How many threads per block ? At most 256, but no more threads than
	 * there are entries to process per block. */
	unsigned nthread_per_block = STARPU_MIN(MAXTHREADSPERBLOCK, (nx / nblocks));

	/* each entry of per_block_cnt contains the number of successful shots
	 * in the corresponding block. */
	monte_carlo<<<nblocks, nthread_per_block, 0, starpu_cuda_get_local_stream()>>>(random_numbers_x, random_numbers_y, nx, per_block_cnt);
	cures = hipGetLastError();
	if (cures != hipSuccess) STARPU_CUDA_REPORT_ERROR(cures);

	/* Note that we do not synchronize between kernel calls because there is an implicit serialization */

	/* compute the total number of successful shots by adding the elements
	 * of the per_block_cnt array */
	sum_per_block_cnt<<<1, nblocks, 0, starpu_cuda_get_local_stream()>>>(per_block_cnt, cnt);
	cures = hipGetLastError();
	if (cures != hipSuccess) STARPU_CUDA_REPORT_ERROR(cures);
	cures = hipStreamSynchronize(starpu_cuda_get_local_stream());
	if (cures)
		STARPU_CUDA_REPORT_ERROR(cures);

	hipFree(per_block_cnt);
	hipFree(random_numbers);
}
