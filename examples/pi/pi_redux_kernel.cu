#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/* This counts how many fall inside the circle quarter */

#include <starpu.h>

#define MAXNBLOCKS	128
#define MAXTHREADSPERBLOCK	256

static __global__ void monte_carlo(float *x, float *y, unsigned n, unsigned long *output_cnt)
{
	__shared__ unsigned scnt[MAXTHREADSPERBLOCK];

	/* Do we have a successful shot ? */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;

	const int nthreads = gridDim.x * blockDim.x;

	/* Blank the shared mem buffer */
	if (threadIdx.x < MAXTHREADSPERBLOCK)
		scnt[threadIdx.x] = 0;

	__syncthreads();
	int ind;
	for (ind = tid; ind < n; ind += nthreads)
	{
		float xval = (2.0f * x[ind] - 1.0f);
		float yval = (2.0f * y[ind] - 1.0f);
		float dist = (xval*xval + yval*yval);

		unsigned long success = (dist <= 1.0f)?1:0;

		scnt[threadIdx.x] += success;

	}

	__syncthreads();

	/* Perform a reduction to compute the sum on each thread within that block */

	/* NB: We assume that the number of threads per block is a power of 2 ! */
	unsigned long s;
	for (s = blockDim.x/2; s!=0; s>>=1)
	{
		if (threadIdx.x < s)
			scnt[threadIdx.x] += scnt[threadIdx.x + s];

		__syncthreads();
	}

	/* report the number of successful shots in the block */
	if (threadIdx.x == 0)
		output_cnt[blockIdx.x] = scnt[0];

	__syncthreads();
}

static __global__ void sum_per_block_cnt(unsigned long *output_cnt, unsigned long *cnt)
{
	__shared__ unsigned long accumulator[MAXNBLOCKS];

	unsigned i;

	/* Load the values from global mem */
	for (i = 0; i < blockDim.x; i++)
		accumulator[i] = output_cnt[i];

	__syncthreads();

	/* Perform a reduction in shared memory */
	unsigned s;
	for (s = blockDim.x/2; s!=0; s>>=1)
	{
		if (threadIdx.x < s)
			accumulator[threadIdx.x] += accumulator[threadIdx.x + s];

		__syncthreads();
	}

	/* Save the result in global memory */
	if (threadIdx.x == 0)
		*cnt = *cnt + accumulator[0];
}

extern "C" void pi_redux_cuda_kernel(float *x, float *y, unsigned n, unsigned long *shot_cnt)
{
	hipError_t cures;

	/* How many blocks do we use ? */
	unsigned nblocks = 128; // TODO
	STARPU_ASSERT(nblocks <= MAXNBLOCKS);
	STARPU_ASSERT((n % nblocks) == 0);

	unsigned long *per_block_cnt;
	hipMalloc((void **)&per_block_cnt, nblocks*sizeof(unsigned long));

	/* How many threads per block ? At most 256, but no more threads than
	 * there are entries to process per block. */
	unsigned nthread_per_block = STARPU_MIN(MAXTHREADSPERBLOCK, (n / nblocks));

	/* each entry of per_block_cnt contains the number of successful shots
	 * in the corresponding block. */
	monte_carlo<<<nblocks, nthread_per_block, 0, starpu_cuda_get_local_stream()>>>(x, y, n, per_block_cnt);
	cures = hipGetLastError();
	if (cures != hipSuccess) STARPU_CUDA_REPORT_ERROR(cures);

	/* Note that we do not synchronize between kernel calls because there is an implicit serialization */

	/* compute the total number of successful shots by adding the elements
	 * of the per_block_cnt array */
	sum_per_block_cnt<<<1, nblocks, 0, starpu_cuda_get_local_stream()>>>(per_block_cnt, shot_cnt);
	cures = hipGetLastError();
	if (cures != hipSuccess) STARPU_CUDA_REPORT_ERROR(cures);
	cures = hipStreamSynchronize(starpu_cuda_get_local_stream());
	if (cures)
		STARPU_CUDA_REPORT_ERROR(cures);

	hipFree(per_block_cnt);
}
