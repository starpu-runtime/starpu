#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2008-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/* CUDA kernel for SPMV */

#include <starpu.h>

#define MIN(a,b)	((a)<(b)?(a):(b))

extern "C" __global__ void spmv_kernel(uint32_t nnz, uint32_t nrow, float *nzval, uint32_t *colind, uint32_t *rowptr,
				       uint32_t firstentry, uint32_t elemsize,
				       float *vecin, uint32_t nx_in, uint32_t elemsize1, float * vecout, uint32_t nx_out, uint32_t elemsize2)
{
	/* only one dimension is used here */
	unsigned nthreads = gridDim.x*blockDim.x;
	unsigned threadid = threadIdx.x + blockIdx.x*blockDim.x;

	unsigned rowstart = threadid * ((nrow + (nthreads - 1))/nthreads);
	unsigned rowend = MIN(nrow, (threadid+1) * ((nrow + (nthreads - 1))/nthreads));

	unsigned row;
	for (row = rowstart; row < rowend; row++)
	{
		float tmp = 0.0f;
		unsigned index;

		unsigned firstindex = rowptr[row] - firstentry;
		unsigned lastindex = rowptr[row+1] - firstentry;

		for (index = firstindex; index < lastindex; index++)
		{
			tmp += nzval[index]*vecin[colind[index]];
		}

		vecout[row] = tmp;
	}
}

extern "C" __global__ void spmv_kernel_3(uint32_t nnz, uint32_t nrow, float *nzval, uint32_t *colind, uint32_t *rowptr,
					 uint32_t firstentry,
					 float *vecin, uint32_t nx_in, float * vecout, uint32_t nx_out)
{
	/* only one dimension is used here */
	unsigned block_rowstart = blockIdx.x*( (nrow + gridDim.x - 1)/gridDim.x );
	unsigned block_rowend = MIN((blockIdx.x+1)*( (nrow + gridDim.x - 1)/gridDim.x ), nrow);

	unsigned row;
	for (row = block_rowstart + threadIdx.x; row < block_rowend; row+=blockDim.x)
	{
		float tmp = 0.0f;
		unsigned index;

		unsigned firstindex = rowptr[row] - firstentry;
		unsigned lastindex = rowptr[row+1] - firstentry;

		for (index = firstindex; index < lastindex; index++)
		{
			tmp += nzval[index]*vecin[colind[index]];
		}

		vecout[row] = tmp;
	}


}

extern "C" void spmv_kernel_cuda(void *descr[], void *args)
{
	uint32_t nnz = STARPU_CSR_GET_NNZ(descr[0]);
	uint32_t nrow = STARPU_CSR_GET_NROW(descr[0]);
	float *nzval = (float *)STARPU_CSR_GET_NZVAL(descr[0]);
	uint32_t *colind = STARPU_CSR_GET_COLIND(descr[0]);
	uint32_t *rowptr = STARPU_CSR_GET_ROWPTR(descr[0]);
	uint32_t firstentry = STARPU_CSR_GET_FIRSTENTRY(descr[0]);

	float *vecin = (float *)STARPU_VECTOR_GET_PTR(descr[1]);
	uint32_t nx_in = STARPU_VECTOR_GET_NX(descr[1]);

	float *vecout = (float *)STARPU_VECTOR_GET_PTR(descr[2]);
	uint32_t nx_out = STARPU_VECTOR_GET_NX(descr[2]);

	dim3 dimBlock(8, 1);
	dim3 dimGrid(512, 1);

	spmv_kernel_3<<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>>
		(nnz, nrow, nzval, colind, rowptr, firstentry, vecin, nx_in, vecout, nx_out);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
