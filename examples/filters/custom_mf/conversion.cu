#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>
#include "custom_types.h"
#include "custom_interface.h"

static __global__ void custom_cuda(struct point *aop,
				unsigned n,
				float *x,
				float *y)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
	{
		x[i] = aop[i].x;
		y[i] = aop[i].y;
	}
}

extern "C" void cpu_to_cuda_cuda_func(void *buffers[], void *_args)
{
	(void) _args;

	unsigned int n = CUSTOM_GET_NX(buffers[0]);
	float *x = (float*) CUSTOM_GET_X_PTR(buffers[0]);
	float *y = (float*) CUSTOM_GET_Y_PTR(buffers[0]);

	struct point *aop;
	aop = (struct point *) CUSTOM_GET_CPU_PTR(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;
        custom_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>>(aop, n, x, y);
        hipError_t status = hipGetLastError();
        if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
