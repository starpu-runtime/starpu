#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#define _externC extern "C"
#include "stencil.h"

/* Heart of the stencil computation: compute a new state from an old one. */

extern "C" __global__ void cuda_life_update(int bz, const TYPE *old, TYPE *newp, int nx, int ny, int nz, int ldy, int ldz, int iter)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned idy = threadIdx.y + blockIdx.y * blockDim.y;
	//unsigned idz = threadIdx.z + blockIdx.z * blockDim.z;
	unsigned idz = 0;
	unsigned stepx = blockDim.x * gridDim.x;
	unsigned stepy = blockDim.y * gridDim.y;
	//unsigned stepz = blockDim.z * gridDim.z;
	unsigned stepz = 1;
	unsigned x, y, z;
	unsigned num, alive;

	for (z = iter + idz; z < nz - iter; z += stepz)
		for (y = K + idy; y < ny - K; y += stepy)
		{
			for (x = K + idx; x < nx - K; x += stepx)
			{
				unsigned index = x + y*ldy + z*ldz;
				num = 0
                                        + old[index+1*ldy+0*ldz]
                                        + old[index+1*ldy+1*ldz]
                                        + old[index+0*ldy+1*ldz]
                                        + old[index-1*ldy+1*ldz]
                                        + old[index-1*ldy+0*ldz]
                                        + old[index-1*ldy-1*ldz]
                                        + old[index+0*ldy-1*ldz]
                                        + old[index+1*ldy-1*ldz]
					;
				alive = old[index];
				alive = (alive && num == 2) || num == 3;
				newp[index] = alive;
			}
		}
}

extern "C" void cuda_life_update_host(int bz, const TYPE *old, TYPE *newp, int nx, int ny, int nz, int ldy, int ldz, int iter)
{
	unsigned max_parallelism = 512;
	unsigned threads_per_dim_x = max_parallelism;
	while (threads_per_dim_x / 2 >= nx)
		threads_per_dim_x /= 2;
	unsigned threads_per_dim_y = max_parallelism / threads_per_dim_x;
	while (threads_per_dim_y / 2 >= ny)
		threads_per_dim_y /= 2;
#if 0
	unsigned threads_per_dim_z = 4;
	dim3 dimBlock(threads_per_dim_x, threads_per_dim_y, threads_per_dim_z);
	dim3 dimGrid(nx / threads_per_dim_x, ny / threads_per_dim_y, nz / threads_per_dim_z);
#else
	dim3 dimBlock(threads_per_dim_x, threads_per_dim_y);
	dim3 dimGrid((nx + threads_per_dim_x-1) / threads_per_dim_x, (ny + threads_per_dim_y-1) / threads_per_dim_y);
#endif
	cuda_life_update <<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>> (bz, old, newp, nx, ny, nz, ldy, ldz, iter);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
