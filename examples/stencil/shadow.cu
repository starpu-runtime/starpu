#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#define _externC extern "C"
#include "stencil.h"

/* Perform replication of data on X and Y edges, to fold the domain on 
   itself through mere replication of the source state. */

extern "C" __global__ void cuda_shadow( int bz, TYPE *ptr, int nx, int ny, int nz, int ldy, int ldz, int i)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned idy = threadIdx.y + blockIdx.y * blockDim.y;
	//unsigned idz = threadIdx.z + blockIdx.z * blockDim.z;
	unsigned idz = 0;
	unsigned stepx = blockDim.x * gridDim.x;
	unsigned stepy = blockDim.y * gridDim.y;
	//unsigned stepz = blockDim.z * gridDim.z;
	unsigned stepz = 1;
	unsigned x, y, z;

#include "shadow.h"
}

extern "C" void cuda_shadow_host(int bz, TYPE *ptr, int nx, int ny, int nz, int ldy, int ldz, int i)
{
	unsigned max_parallelism = 512;
	unsigned threads_per_dim_x = max_parallelism;
	while (threads_per_dim_x / 2 >= nx)
		threads_per_dim_x /= 2;
	unsigned threads_per_dim_y = max_parallelism / threads_per_dim_x;
	while (threads_per_dim_y / 2 >= ny)
		threads_per_dim_y /= 2;
#if 0
	unsigned threads_per_dim_z = 4;
	dim3 dimBlock(threads_per_dim_x, threads_per_dim_y, threads_per_dim_z);
	dim3 dimGrid(nx / threads_per_dim_x, ny / threads_per_dim_y, nz / threads_per_dim_z);
#else
	dim3 dimBlock(threads_per_dim_x, threads_per_dim_y);
	dim3 dimGrid((nx + threads_per_dim_x-1) / threads_per_dim_x, (ny + threads_per_dim_y-1) / threads_per_dim_y);
#endif
	cuda_shadow <<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>> (bz, ptr, nx, ny, nz, ldy, ldz, i);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
