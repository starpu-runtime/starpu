#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>
#include "multiformat_types.h"

#define FPRINTF(ofile, fmt, ...) do { if (!getenv("STARPU_SSILENT")) {fprintf(ofile, fmt, ## __VA_ARGS__); }} while(0)

static __global__ void multiformat_cuda(struct struct_of_arrays *soa, unsigned n)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
		soa->x[i] *= soa->y[i];
}

extern "C" void multiformat_scal_cuda_func(void *buffers[], void *_args)
{
	(void) _args;

	FPRINTF(stderr, "Running the cuda kernel (%s)\n", __starpu_func__);
	unsigned int n = STARPU_MULTIFORMAT_GET_NX(buffers[0]);
	struct struct_of_arrays *soa;

	soa = (struct struct_of_arrays *) STARPU_MULTIFORMAT_GET_CUDA_PTR(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;
        multiformat_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>>(soa, n);
        hipError_t status = hipGetLastError();
        if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);

	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
