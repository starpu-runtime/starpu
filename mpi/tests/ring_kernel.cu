#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2009-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void cuda_incrementer(int *token)
{
	(*token)++;
}

extern "C" void increment_cuda(void *descr[], void *_args)
{
	(void) _args;
	int *tokenptr = (int *)STARPU_VECTOR_GET_PTR(descr[0]);

	cuda_incrementer<<<1,1, 0, starpu_cuda_get_local_stream()>>>(tokenptr);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
