#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2009-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

extern "C" __global__
void long_kernel(unsigned long niters)
{
	unsigned long i;
	for (i = 0; i < niters; i++)
		__syncthreads();
}

extern "C" void long_kernel_cuda(unsigned long niters)
{
	dim3 dimBlock(1,1);
	dim3 dimGrid(1,1);
	long_kernel<<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>>(niters);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
