#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "multiformat_types.h"
#include "../../../helper.h"

static __global__ void cpu_to_cuda_cuda(struct point *src,
	struct struct_of_arrays *dst, unsigned n)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
	{
		dst->x[i] = src[i].x;
		dst->y[i] = src[i].y;
	}

}

extern "C" void cpu_to_cuda_cuda_func(void *buffers[], void *_args)
{
	FPRINTF(stderr, "Entering %s\n", __starpu_func__);
	struct point *src;
	struct struct_of_arrays *dst;

	src = (struct point *) STARPU_MULTIFORMAT_GET_CPU_PTR(buffers[0]);
	dst = (struct struct_of_arrays *) STARPU_MULTIFORMAT_GET_CUDA_PTR(buffers[0]);

	int n = STARPU_MULTIFORMAT_GET_NX(buffers[0]);

	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

        cpu_to_cuda_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>>(src, dst, n);
        hipError_t status = hipGetLastError();
        if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
