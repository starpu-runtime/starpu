#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "multiformat_types.h"
#include "../test_interfaces.h"
#include "../../../helper.h"

extern struct test_config multiformat_config;

static __global__ void multiformat_cuda(struct struct_of_arrays *soa, unsigned n,
					int *err, int factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= n)
		return;

	if (soa->x[i] != i * factor || soa->y[i] != i * factor)
	{
		*err = 1;
	}
	else
	{
		soa->x[i] = -soa->x[i];
		soa->y[i] = -soa->y[i];
	}
}

extern "C" void test_multiformat_cuda_func(void *buffers[], void *args)
{
	FPRINTF(stderr, "Entering %s\n", __starpu_func__);
	int factor;
	int *ret;
	hipError_t error;
	unsigned int n = STARPU_MULTIFORMAT_GET_NX(buffers[0]);
	struct struct_of_arrays *soa;

	soa = (struct struct_of_arrays *) STARPU_MULTIFORMAT_GET_CUDA_PTR(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;
	factor = *(int *) args;

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret,
			   &multiformat_config.copy_failed,
			   sizeof(int),
			   hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

        multiformat_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>>(soa, n, ret, factor);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(&multiformat_config.copy_failed,
			   ret,
			   sizeof(int),
			   hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
