#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config csr_config;

__global__ void csr_cuda(int *nzval, uint32_t nnz, int *err, int factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nnz)
		return;

	if (nzval[i] != (i+1)*factor)
		*err = 1;
	else
		nzval[i] = -nzval[i];
}

extern "C" void test_csr_cuda_func(void *buffers[], void *args)
{
	int factor;
	int *ret;
	int *val;
	hipError_t error;
	uint32_t nnz = STARPU_CSR_GET_NNZ(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (nnz + threads_per_block-1) / threads_per_block;

	factor = *(int *) args;
	val = (int *) STARPU_CSR_GET_NZVAL(buffers[0]);

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret,
			   &csr_config.copy_failed,
			   sizeof(int),
			   hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

        csr_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>> (val, nnz, ret, factor);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(&csr_config.copy_failed,
			   ret,
			   sizeof(int),
			   hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
