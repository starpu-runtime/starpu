#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config variable_config;

static __global__ void variable_cuda(int *val, int *err, int factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i > 0)
		return;

	if (*val != 42 * factor)
		*err = 1;
	else
		*val *= -1;
}

extern "C" void test_variable_cuda_func(void *buffers[], void *args)
{
	hipError_t error;
	int *ret;

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret,
			   &variable_config.copy_failed,
			   sizeof(int),
			   hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

        int *val = (int *)STARPU_VARIABLE_GET_PTR(buffers[0]);
	int factor = *(int*) args;

	unsigned threads_per_block = 64;
	unsigned nblocks = 1;

        variable_cuda<<<nblocks,threads_per_block,0,starpu_cuda_get_local_stream()>>>(val, ret, factor);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);
	error = hipMemcpyAsync(&variable_config.copy_failed,
			   ret,
			   sizeof(int),
			   hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
