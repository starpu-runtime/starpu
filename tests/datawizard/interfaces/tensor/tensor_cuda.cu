#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2011-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config tensor_config;

static __global__ void tensor_cuda(int *tensor,
				  int nx, int ny, int nz, int nt,
				  unsigned ldy, unsigned ldz, unsigned ldt,
				  int factor, int *err)
{
        int i, j, k, l;
	int val = 0;

        for (l = 0; l < nt ;l++)
	{
	    for (k = 0; k < nz ;k++)
	    {
                for (j = 0; j < ny ;j++)
		{
                        for(i = 0; i < nx ;i++)
			{
				if (tensor[(l*ldt)+(k*ldz)+(j*ldy)+i] != factor * val)
				{
					*err = 1;
					return;
				}
				else
				{
					tensor[(l*ldt)+(k*ldz)+(j*ldy)+i] *= -1;
					val++;
				}
			}
                }
	    }
        }
}

extern "C" void test_tensor_cuda_func(void *buffers[], void *args)
{
	hipError_t error;
	int *ret;

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret, &tensor_config.copy_failed, sizeof(int), hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	int nx = STARPU_TENSOR_GET_NX(buffers[0]);
	int ny = STARPU_TENSOR_GET_NY(buffers[0]);
	int nz = STARPU_TENSOR_GET_NZ(buffers[0]);
	int nt = STARPU_TENSOR_GET_NT(buffers[0]);
        unsigned ldy = STARPU_TENSOR_GET_LDY(buffers[0]);
        unsigned ldz = STARPU_TENSOR_GET_LDZ(buffers[0]);
        unsigned ldt = STARPU_TENSOR_GET_LDT(buffers[0]);
	int *tensor = (int *) STARPU_TENSOR_GET_PTR(buffers[0]);
	int factor = *(int*) args;

        tensor_cuda<<<1,1, 0, starpu_cuda_get_local_stream()>>>
		(tensor, nx, ny, nz, nt, ldy, ldz, ldt, factor, ret);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);
	error = hipMemcpyAsync(&tensor_config.copy_failed, ret, sizeof(int), hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
