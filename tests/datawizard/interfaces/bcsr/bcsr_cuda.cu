#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config bcsr_config;

__global__ void bcsr_cuda(int *nzval, uint32_t nnz, int *err, int factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nnz)
		return;

	if (nzval[i] != i*factor)
		*err = 1;
	else
		nzval[i] = -nzval[i];
}

extern "C" void test_bcsr_cuda_func(void *buffers[], void *args)
{
	int factor;
	int *ret;
	int *val;
	hipError_t error;
	uint32_t nnz = STARPU_BCSR_GET_NNZ(buffers[0]);
 	uint32_t r   = ((struct starpu_bcsr_interface *)buffers[0])->r;
 	uint32_t c   = ((struct starpu_bcsr_interface *)buffers[0])->c;
	nnz *= (r*c);
	unsigned threads_per_block = 64;
	unsigned nblocks = (nnz + threads_per_block-1) / threads_per_block;

	factor = *(int *) args;
	val = (int *) STARPU_BCSR_GET_NZVAL(buffers[0]);

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret,
			   &bcsr_config.copy_failed,
			   sizeof(int),
			   hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

        bcsr_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>>
		(val, nnz, ret, factor);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(&bcsr_config.copy_failed,
			   ret,
			   sizeof(int),
			   hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
