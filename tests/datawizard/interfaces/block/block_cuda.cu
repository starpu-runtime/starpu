#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2011-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config block_config;

static __global__ void block_cuda(int *block,
				  int nx, int ny, int nz,
				  unsigned ldy, unsigned ldz,
				  float factor, int *err)
{
        int i, j, k;
	int val = 0;

        for (k = 0; k < nz ;k++)
	{
                for (j = 0; j < ny ;j++)
		{
                        for(i = 0; i < nx ;i++)
			{
				if (block[(k*ldz)+(j*ldy)+i] != factor * val)
				{
					*err = 1;
					return;
				}
				else
				{
					block[(k*ldz)+(j*ldy)+i] *= -1;
					val++;
				}
			}
                }
        }
}

extern "C" void test_block_cuda_func(void *buffers[], void *args)
{
	hipError_t error;
	int *ret;

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret, &block_config.copy_failed, sizeof(int), hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	int nx = STARPU_BLOCK_GET_NX(buffers[0]);
	int ny = STARPU_BLOCK_GET_NY(buffers[0]);
	int nz = STARPU_BLOCK_GET_NZ(buffers[0]);
        unsigned ldy = STARPU_BLOCK_GET_LDY(buffers[0]);
        unsigned ldz = STARPU_BLOCK_GET_LDZ(buffers[0]);
	int *block = (int *) STARPU_BLOCK_GET_PTR(buffers[0]);
	int factor = *(int*) args;

        block_cuda<<<1,1, 0, starpu_cuda_get_local_stream()>>>
		(block, nx, ny, nz, ldy, ldz, factor, ret);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);
	error = hipMemcpyAsync(&block_config.copy_failed, ret, sizeof(int), hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
