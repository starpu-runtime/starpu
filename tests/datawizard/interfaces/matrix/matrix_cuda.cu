#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config matrix_config;

__global__ void matrix_cuda(int *val, unsigned n, int *err, int factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= n)
		return;

	if (val[i] != i*factor)
		*err = 1;
	else
		val[i] = -val[i];
}

extern "C" void test_matrix_cuda_func(void *buffers[], void *args)
{
	int factor;
	int *ret;
	int *val;
	hipError_t error;
	unsigned int nx, ny, n;

	nx = STARPU_MATRIX_GET_NX(buffers[0]);
	ny = STARPU_MATRIX_GET_NY(buffers[0]);
	n = nx * ny;
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;
	factor = *(int *) args;
	val = (int *) STARPU_MATRIX_GET_PTR(buffers[0]);

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(ret,
			   &matrix_config.copy_failed,
			   sizeof(int),
			   hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

        matrix_cuda<<<nblocks,threads_per_block,2,starpu_cuda_get_local_stream()>>>(val, n, ret, factor);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);

	error = hipMemcpyAsync(&matrix_config.copy_failed,
			   ret,
			   sizeof(int),
			   hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		STARPU_CUDA_REPORT_ERROR(error);

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
