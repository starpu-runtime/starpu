#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */
#include <starpu.h>
#include "../test_interfaces.h"

extern struct test_config vector_config;

__global__ void framework_cuda(int *val, unsigned n, int *err, int factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= n)
		return;

	if (val[i] != i*factor)
		*err = 1;
	else
		val[i] = -val[i];
}

extern "C" void test_vector_cuda_func(void *buffers[], void *args)
{
	hipError_t error;
	int *ret;

	error = hipMalloc(&ret, sizeof(int));
	if (error != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed...\n");
		return;
	}

	error = hipMemcpyAsync(ret, &vector_config.copy_failed, sizeof(int), hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
		return;

        unsigned n = STARPU_VECTOR_GET_NX(buffers[0]);
        int *val = (int *)STARPU_VECTOR_GET_PTR(buffers[0]);
	int factor = *(int*) args;

	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

        framework_cuda<<<nblocks,threads_per_block,0,starpu_cuda_get_local_stream()>>>(val, n, ret, factor);
	error = hipGetLastError();
	if (error != hipSuccess) STARPU_CUDA_REPORT_ERROR(error);
	error = hipMemcpyAsync(&vector_config.copy_failed, ret, sizeof(int), hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	if (error != hipSuccess)
	{
		return;
	}

	hipFree(ret);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
