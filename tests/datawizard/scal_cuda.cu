#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void vector_mult_cuda(unsigned *val, unsigned n)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
               val[i] *= 2;
}

extern "C" void scal_func_cuda(void *buffers[], void *_args)
{
        unsigned n = STARPU_VECTOR_GET_NX(buffers[0]);
        unsigned *val = (unsigned *)STARPU_VECTOR_GET_PTR(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

        vector_mult_cuda<<<nblocks,threads_per_block,0,starpu_cuda_get_local_stream()>>>(val, n);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
