#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <stdio.h>
#include <starpu.h>

#define MAXNBLOCKS		32
#define MAXTHREADSPERBLOCK	128

static __global__ void increment_vector(unsigned *v, unsigned *tmp, int nx)
{
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const int nthreads = gridDim.x * blockDim.x;

	int i;
	for (i = tid; i < nx; i += nthreads)
	{
		v[i] = tmp[i] + 1;
	}
}

extern "C" void cuda_f(void *descr[], void *_args)
{
	unsigned *v = (unsigned *)STARPU_VECTOR_GET_PTR(descr[0]);
	unsigned *tmp = (unsigned *)STARPU_VECTOR_GET_PTR(descr[1]);

	unsigned nx = STARPU_VECTOR_GET_NX(descr[0]);
	size_t elemsize = STARPU_VECTOR_GET_ELEMSIZE(descr[0]);

	hipMemcpyAsync(tmp, v, nx*elemsize, hipMemcpyDeviceToDevice, starpu_cuda_get_local_stream());

	unsigned nblocks = 128;
	unsigned nthread_per_block = STARPU_MIN(MAXTHREADSPERBLOCK, (nx / nblocks));

	increment_vector<<<nblocks, nthread_per_block, 0, starpu_cuda_get_local_stream()>>>(v, tmp, nx);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}
