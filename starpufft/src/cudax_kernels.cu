#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2009-2025  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#define _externC extern "C"
#include "cudax_kernels.h"

/* Note: these assume that the sizes are powers of two */

#define VARS_1d \
	unsigned start = threadIdx.x + blockIdx.x * blockDim.x; \
	unsigned numthreads = blockDim.x * gridDim.x;

#define DISTRIB_1d(n, func,args) \
	unsigned threads_per_block = 128; \
\
	if (n < threads_per_block) \
	{			   \
		dim3 dimGrid(n); \
		func <<<dimGrid, 1, 0, starpu_cuda_get_local_stream()>>> args; \
		hipError_t status = hipGetLastError(); \
		if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status); \
	} 					\
	else 					\
	{				     \
		dim3 dimGrid(n / threads_per_block); \
		dim3 dimBlock(threads_per_block); \
		func <<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>> args; \
		hipError_t status = hipGetLastError(); \
		if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status); \
	} \
	hipStreamSynchronize(starpu_cuda_get_local_stream()); \

extern "C" __global__ void STARPUFFT(cuda_twist1_1d)(const _cuComplex *in, _cuComplex *twisted1, unsigned i, unsigned n1, unsigned n2)
{
	unsigned j;
	VARS_1d
	unsigned end = n2;

	for (j = start; j < end; j += numthreads)
		twisted1[j] = in[i+j*n1];
}

extern "C" void STARPUFFT(cuda_twist1_1d_host)(const _cuComplex *in, _cuComplex *twisted1, unsigned i, unsigned n1, unsigned n2)
{
	DISTRIB_1d(n2, STARPUFFT(cuda_twist1_1d), (in, twisted1, i, n1, n2));
}

extern "C" __global__ void STARPUFFT(cuda_twiddle_1d)(_cuComplex * out, const _cuComplex * roots, unsigned n, unsigned i)
{
	unsigned j;
	VARS_1d
	unsigned end = n;

	for (j = start; j < end; j += numthreads)
		out[j] = _cuCmul(out[j], roots[i*j]);
	return;
}

extern "C" void STARPUFFT(cuda_twiddle_1d_host)(_cuComplex *out, const _cuComplex *roots, unsigned n, unsigned i)
{
	DISTRIB_1d(n, STARPUFFT(cuda_twiddle_1d), (out, roots, n, i));
}

#define VARS_2d \
	unsigned startx = threadIdx.x + blockIdx.x * blockDim.x; \
	unsigned starty = threadIdx.y + blockIdx.y * blockDim.y; \
	unsigned numthreadsx = blockDim.x * gridDim.x; \
	unsigned numthreadsy = blockDim.y * gridDim.y;

/* FIXME: introduce threads_per_dim_n / m instead */
#define DISTRIB_2d(n, m, func, args) \
	unsigned threads_per_dim = 16; \
	if (n < threads_per_dim) \
	{				   \
		if (m < threads_per_dim) \
		{			    \
			dim3 dimGrid(n, m); \
			func <<<dimGrid, 1, 0, starpu_cuda_get_local_stream()>>> args; \
			hipError_t status = hipGetLastError(); \
			if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status); \
		} \
		else \
		{					      \
			dim3 dimGrid(1, m / threads_per_dim); \
			dim3 dimBlock(n, threads_per_dim); \
			func <<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>> args; \
			hipError_t status = hipGetLastError(); \
			if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status); \
		} \
	} \
	else \
	{				   \
		if (m < threads_per_dim) \
		{					      \
			dim3 dimGrid(n / threads_per_dim, 1); \
			dim3 dimBlock(threads_per_dim, m); \
			func <<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>> args; \
			hipError_t status = hipGetLastError(); \
			if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status); \
		} \
		else \
		{							\
			dim3 dimGrid(n / threads_per_dim, m / threads_per_dim); \
			dim3 dimBlock(threads_per_dim, threads_per_dim); \
			func <<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>> args; \
			hipError_t status = hipGetLastError(); \
			if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status); \
		} \
	} \
	hipStreamSynchronize(starpu_cuda_get_local_stream()); \

extern "C" __global__ void STARPUFFT(cuda_twist1_2d)(const _cuComplex *in, _cuComplex *twisted1, unsigned i, unsigned j, unsigned n1, unsigned n2, unsigned m1, unsigned m2)
{
	unsigned k, l;
	VARS_2d
	unsigned endx = n2;
	unsigned endy = m2;
	unsigned m = m1*m2;

	for (k = startx; k < endx; k += numthreadsx)
		for (l = starty; l < endy; l += numthreadsy)
			twisted1[k*m2+l] = in[i*m+j+k*m*n1+l*m1];
}

extern "C" void STARPUFFT(cuda_twist1_2d_host)(const _cuComplex *in, _cuComplex *twisted1, unsigned i, unsigned j, unsigned n1, unsigned n2, unsigned m1, unsigned m2)
{
	DISTRIB_2d(n2, m2, STARPUFFT(cuda_twist1_2d), (in, twisted1, i, j, n1, n2, m1, m2));
}

extern "C" __global__ void STARPUFFT(cuda_twiddle_2d)(_cuComplex * out, const _cuComplex * roots0, const _cuComplex * roots1, unsigned n2, unsigned m2, unsigned i, unsigned j)
{
	unsigned k, l;
	VARS_2d
	unsigned endx = n2;
	unsigned endy = m2;

	for (k = startx; k < endx ; k += numthreadsx)
		for (l = starty; l < endy ; l += numthreadsy)
			out[k*m2 + l] = _cuCmul(_cuCmul(out[k*m2 + l], roots0[i*k]), roots1[j*l]);
	return;
}

extern "C" void STARPUFFT(cuda_twiddle_2d_host)(_cuComplex *out, const _cuComplex *roots0, const _cuComplex *roots1, unsigned n2, unsigned m2, unsigned i, unsigned j)
{
	DISTRIB_2d(n2, m2, STARPUFFT(cuda_twiddle_2d), (out, roots0, roots1, n2, m2, i, j));
}
